#include "hip/hip_runtime.h"

#include <iostream>

#include "file1.h"
#include "file2.h"

int file2_launch_kernel(int x);

result_type_dynamic __device__ file2_func(int x);
static __global__ void main_kernel(result_type_dynamic& r, int x)
{
  // call function that was not device linked to us, this will cause
  // a runtime failure of "invalid device function"
  r = file2_func(x);
}

int main_launch_kernel(int x)
{
  result_type_dynamic r;
  main_kernel<<<1, 1>>>(r, x);
  return r.sum;
}

int choose_cuda_device()
{
  int nDevices = 0;
  hipError_t err = hipGetDeviceCount(&nDevices);
  if (err != hipSuccess) {
    std::cerr << "Failed to retrieve the number of CUDA enabled devices"
              << std::endl;
    return 1;
  }
  for (int i = 0; i < nDevices; ++i) {
    hipDeviceProp_t prop;
    hipError_t err = hipGetDeviceProperties(&prop, i);
    if (err != hipSuccess) {
      std::cerr << "Could not retrieve properties from CUDA device " << i
                << std::endl;
      return 1;
    }
    std::cout << "prop.major: " << prop.major << std::endl;
    if (prop.major >= 3) {
      err = hipSetDevice(i);
      if (err != hipSuccess) {
        std::cout << "Could not select CUDA device " << i << std::endl;
      } else {
        return 0;
      }
    }
  }

  std::cout << "Could not find a CUDA enabled card supporting compute >=3.0"
            << std::endl;

  return 1;
}

int main(int argc, char** argv)
{
  int ret = choose_cuda_device();
  if (ret) {
    return 0;
  }

  main_launch_kernel(1);
  hipError_t err = hipGetLastError();
  if (err == hipSuccess) {
    // This kernel launch should fail as the file2_func was device linked
    // into the static library and is not usable by the executable
    std::cerr << "main_launch_kernel: kernel launch should have failed"
              << std::endl;
    return 1;
  }

  return 0;
}
