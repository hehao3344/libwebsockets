#include "hip/hip_runtime.h"

#include "file2.h"

result_type __device__ file1_func(int x);

result_type_dynamic __device__ file2_func(int x)
{
  const result_type r = file1_func(x);
  const result_type_dynamic rd{ r.input, r.sum, true };
  return rd;
}

static __global__ void file2_kernel(result_type_dynamic& r, int x)
{
  // call static_func which is a method that is defined in the
  // static library that is always out of date
  r = file2_func(x);
}

int file2_launch_kernel(int x)
{
  result_type_dynamic r;
  file2_kernel<<<1, 1>>>(r, x);
  return r.sum;
}
